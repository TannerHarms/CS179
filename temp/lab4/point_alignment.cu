/**
 * CUDA Point Alignment
 * George Stathopoulos, Jenny Lee, Mary Giambrone, 2019*/ 

#include <cstdio>
#include <stdio.h>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "hip/hip_runtime_api.h"
#include <string>
#include <fstream>

#include "obj_structures.h"

// hip/hip_runtime_api.h contains the error checking macros. note that they're called
// CUDA_CALL, CUBLAS_CALL, and CUSOLVER_CALL instead of the previous names

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char *argv[]) {

    if (argc != 4)
    {
        printf("Usage: ./point_alignment [file1.obj] [file2.obj] [output.obj]\n");
        return 1;
    }

    std::string filename, filename2, output_filename;
    filename = argv[1];
    filename2 = argv[2];
    output_filename = argv[3];

    std::cout << "Aligning " << filename << " with " << filename2 <<  std::endl;
    Object obj1 = read_obj_file(filename);
    std::cout << "Reading " << filename << ", which has " << obj1.vertices.size() << " vertices" << std::endl;
    Object obj2 = read_obj_file(filename2);

    std::cout << "Reading " << filename2 << ", which has " << obj2.vertices.size() << " vertices" << std::endl;
    if (obj1.vertices.size() != obj2.vertices.size())
    {
        printf("Error: number of vertices in the obj files do not match.\n");
        return 1;
    }

    ///////////////////////////////////////////////////////////////////////////
    // Loading in obj into vertex Array
    ///////////////////////////////////////////////////////////////////////////

    int point_dim = 4; // 3 spatial + 1 homogeneous
    int num_points = obj1.vertices.size();

    // in col-major
    float * x1mat = vertex_array_from_obj(obj1);
    float * x2mat = vertex_array_from_obj(obj2);

    ///////////////////////////////////////////////////////////////////////////
    // Point Alignment
    ///////////////////////////////////////////////////////////////////////////

    // TODO: Initialize cublas handle
    hipblasHandle_t handle;
    CUBLAS_CALL(hipblasCreate(&handle));

    float * dev_x1mat;
    float * dev_x2mat;
    float * dev_xx4x4;
    float * dev_x1Tx2;

    // TODO: Allocate device memory and copy over the data onto the device
    // Hint: Use hipblasSetMatrix() for copying

    // Leading dimensions
    int ld1 = num_points;   // leading dimension of x1 column-major
    int ld2 = num_points;   // leading dimension of x2 column-major
    int ld3 = point_dim;   // leading dimension of x1^(T)*x1 column-major
    int ld4 = point_dim;   // leading dimension of x1^(T)*x2 column-major

    // Allocate the memory 
    CUDA_CALL(hipMalloc((void**)&dev_x1mat, ld1 * point_dim * sizeof(float)));        // n by 4
    CUDA_CALL(hipMalloc((void**)&dev_x2mat, ld2 * point_dim * sizeof(float)));        // n by 4
    CUDA_CALL(hipMalloc((void**)&dev_xx4x4, point_dim * point_dim * sizeof(float)));   // 4 by 4
    CUDA_CALL(hipMalloc((void**)&dev_x1Tx2, point_dim * point_dim * sizeof(float)));   // 4 by 4

    CUBLAS_CALL(hipblasSetMatrix(num_points, point_dim, sizeof(float), x1mat, ld1, dev_x1mat, ld1));
    CUBLAS_CALL(hipblasSetMatrix(num_points, point_dim, sizeof(float), x2mat, ld2, dev_x2mat, ld2));
    
    // Now, proceed with the computations necessary to solve for the linear
    // transformation.

    float one = 1;
    float zero = 0;

    // cuBLAS transpose or no transpose operations.
    hipblasOperation_t transOn  = HIPBLAS_OP_T;
    hipblasOperation_t transOff = HIPBLAS_OP_N;

    // TODO: First calculate xx4x4 and x1Tx2
    // Following two calls should correspond to:
    //   xx4x4 = Transpose[x1mat] . x1mat
    // In English...
    //   Simple matrix matrix mulitplication sgemm(handle, transpose condition,transpose condition, output m, output n, 
    //   inner dimension, one = no addition of C afterwards, x1 data, leading dim x1, x1 data, leading dim x1,
    //   no addition of x1Tx1, leading dim x1Tx1).
    CUBLAS_CALL(hipblasSgemm(handle, transOn, transOff, point_dim, point_dim, num_points,
         &one, dev_x1mat, ld1, dev_x1mat, ld1, &zero, dev_xx4x4, ld3));

    //   x1Tx2 = Transpose[x1mat] . x2mat
    CUBLAS_CALL(hipblasSgemm(handle, transOn, transOff, point_dim, point_dim, num_points,
        &one, dev_x1mat, ld1, dev_x2mat, ld2, &zero, dev_x1Tx2, ld4));

    // TODO: Finally, solve the system using LU-factorization! We're solving
    //         xx4x4 . m4x4mat.T = x1Tx2   i.e.   m4x4mat.T = Inverse[xx4x4] . x1Tx2
    //
    //       Factorize xx4x4 into an L and U matrix, ie.  xx4x4 = LU
    //
    //       Then, solve the following two systems at once using cusolver's getrs
    //           L . temp  =  P . x1Tx2
    //       And then then,
    //           U . m4x4mat = temp
    //
    //       Generally, pre-factoring a matrix is a very good strategy when
    //       it is needed for repeated solves.

    // TODO: Make handle for cuSolver
    hipsolverHandle_t solver_handle;
    CUSOLVER_CALL(hipsolverDnCreate(&solver_handle));

    // TODO: Initialize work buffer using hipsolverDnSgetrf_bufferSize
    float * work;
    int Lwork;
    CUSOLVER_CALL(hipsolverDnSgetrf_bufferSize(solver_handle, point_dim, point_dim, dev_xx4x4, point_dim, &Lwork));

    // TODO: compute buffer size and prepare memory
    CUDA_CALL(hipMalloc((void**)&work, Lwork * sizeof(float)));

    // TODO: Initialize memory for pivot array, with a size of point_dim
    int * pivots;
    CUDA_CALL(hipMalloc((void**)&pivots, point_dim * sizeof(int)));   // 4

    int * info;
    CUDA_CALL(hipMalloc((void**)&info, sizeof(int)));   // 1

    // TODO: Now, call the factorizer hipsolverDnSgetrf, using the above initialized data
    CUSOLVER_CALL(hipsolverDnSgetrf(solver_handle, point_dim, point_dim, dev_xx4x4, point_dim, 
        work, pivots, info));

    // TODO: Finally, solve the factorized version using a direct call to hipsolverDnSgetrs
    CUSOLVER_CALL(hipsolverDnSgetrs(solver_handle, transOff, point_dim, point_dim, dev_xx4x4, point_dim, 
        pivots, dev_x1Tx2, point_dim, info));

    // TODO: Destroy the cuSolver handle
    CUSOLVER_CALL(hipsolverDnDestroy(solver_handle));
    CUDA_CALL(hipFree(work));
    CUDA_CALL(hipFree(pivots));
    CUDA_CALL(hipFree(info));

    // TODO: Copy final transformation back to host. Note that at this point
    // the transformation matrix is transposed
    float * out_transformation = (float *)malloc(point_dim * point_dim * sizeof(float));
    CUBLAS_CALL(hipblasGetVector(point_dim * point_dim, sizeof(float), dev_x1Tx2, 1, out_transformation, 1));
    // CUDA_CALL(hipMemcpy(out_transformation, dev_x1Tx2, sizeof(float) * point_dim * point_dim,
    //     hipMemcpyDeviceToHost));

    // TODO: Don't forget to set the bottom row of the final transformation
    //       to [0,0,0,1] (right-most columns of the transposed matrix)
    for (int i = 0; i < 3; i++) {
        out_transformation[IDX2C(i,4,4)] = 0;
    }
    out_transformation[IDX2C(4,4,4)] = 1;

    // Print transformation in row order.
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            std::cout << out_transformation[i * point_dim + j] << " ";
        }
        std::cout << "\n";
    }

    ///////////////////////////////////////////////////////////////////////////
    // Transform point and print output object file
    ///////////////////////////////////////////////////////////////////////////
    
    std::cout << "check 1 " << point_dim << " " << num_points << std::endl;
    // TODO Allocate and Initialize data matrix
    float * dev_pt;
    CUDA_CALL(hipMalloc((void**)&dev_pt, point_dim * num_points * sizeof(float)));        // n by 4
    CUBLAS_CALL(hipblasSetMatrix(num_points, point_dim, sizeof(float), x1mat, num_points, dev_pt, num_points));
        std::cout << "check 1" << std::endl;

    // TODO Allocate and Initialize transformation matrix
    float * dev_trans_mat;
    CUDA_CALL(hipMalloc((void**)&dev_trans_mat, point_dim * point_dim * sizeof(float)));        // 4 by 4
    CUBLAS_CALL(hipblasSetMatrix(point_dim, point_dim, sizeof(float), 
        out_transformation, point_dim, dev_trans_mat, point_dim));
        std::cout << "check 1" << std::endl;

    // TODO Allocate and Initialize transformed points
    float * dev_trans_pt;
    CUDA_CALL(hipMalloc((void**)&dev_trans_pt, point_dim * num_points * sizeof(float)));        // n by 4
    std::cout << "check 1" << std::endl;
    float one_d = 1;
    float zero_d = 0;

    // TODO Transform point matrix
    //          (4x4 trans_mat) . (nx4 pointzx matrix)^T = (4xn transformed points)
    CUBLAS_CALL(hipblasSgemm(handle, transOn, transOn, point_dim, num_points, point_dim,
        &one_d, dev_trans_mat, point_dim, dev_pt, num_points, &zero_d, dev_trans_pt, point_dim));

    std::cout << "check 1" << std::endl;
    // So now dev_trans_pt has shape (4 x n)
    float * trans_pt = (float *)malloc(num_points * point_dim * sizeof(float)); 
    CUDA_CALL(hipMemcpy(trans_pt, dev_trans_pt, sizeof(float) * num_points * point_dim, 
        hipMemcpyDeviceToHost));
    std::cout << "check 1" << std::endl;

    // get Object from transformed vertex matrix
    Object trans_obj = obj_from_vertex_array(trans_pt, num_points, point_dim, obj1);

    // print Object to output file
    std::ofstream obj_file (output_filename);
    print_obj_data(trans_obj, obj_file);

    // free CPU memory
    free(trans_pt);

    ///////////////////////////////////////////////////////////////////////////
    // Free Memory
    ///////////////////////////////////////////////////////////////////////////

    // TODO: Free GPU memory
    CUDA_CALL(hipFree(dev_x1mat));
    CUDA_CALL(hipFree(dev_x2mat));
    CUDA_CALL(hipFree(dev_xx4x4));
    CUDA_CALL(hipFree(dev_x1Tx2));
    CUDA_CALL(hipFree(dev_pt));
    CUDA_CALL(hipFree(dev_trans_mat));
    CUDA_CALL(hipFree(dev_trans_pt));
    
    CUBLAS_CALL(hipblasDestroy(handle));

    // TODO: Free CPU memory
    free(out_transformation);
    free(x1mat);
    free(x2mat);

}

