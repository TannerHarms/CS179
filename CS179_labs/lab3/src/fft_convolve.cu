#include "hip/hip_runtime.h"
/* CUDA blur
 * Kevin Yuh, 2014 */

#include <cstdio>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_convolve.cuh"


/* 
Atomic-max function. You may find it useful for normalization.

We haven't really talked about this yet, but __device__ functions not
only are run on the GPU, but are called from within a kernel.

Source: 
http://stackoverflow.com/questions/17399119/
cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
*/
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}



__global__
void
cudaProdScaleKernel(const hipfftComplex *raw_data, const hipfftComplex *impulse_v, 
    hipfftComplex *out_data, int padded_length) {
    
    // It makes sense to just use one dimension here.  Also, problem oriented this way
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Need to scale by record length due to FFT implementation in cuFFT.  
    float inverse_padded_length = 1.0 / padded_length;

    while (thread_index < padded_length) {
        // Do complex multiplication
        // Store temporary variables

        float a = raw_data[thread_index].x;
        float b = raw_data[thread_index].y;
        float c = impulse_v[thread_index].x;
        float d = impulse_v[thread_index].y;

        hipfftComplex output;

        // Update the output
        output.x = ((a * c) - (b * d)) * inverse_padded_length;
        output.y = ((a * d) + (b * c)) * inverse_padded_length;

        out_data[thread_index] = output;

        // Update the grid stride index
        thread_index += blockDim.x * gridDim.x;
    }

    /* TODO: Implement the point-wise multiplication and scaling for the
    FFT'd input and impulse response. 

    Recall that these are complex numbers, so you'll need to use the
    appropriate rule for multiplying them. 

    Also remember to scale by the padded length of the signal
    (see the notes for Question 1).

    As in Assignment 1 and Week 1, remember to make your implementation
    resilient to varying numbers of threads.

    */
}

__global__
void
cudaMaximumKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the maximum-finding.

    There are many ways to do this reduction, and some methods
    have much better performance than others. 

    For this section: Please explain your approach to the reduction,
    including why you chose the optimizations you did
    (especially as they relate to GPU hardware).

    You'll likely find the above atomicMax function helpful.
    (CUDA's atomicMax function doesn't work for floating-point values.)
    It's based on two principles:
        1) From Week 2, any atomic function can be implemented using
        atomic compare-and-swap.
        2) One can "represent" floating-point values as integers in
        a way that preserves comparison, if the sign of the two
        values is the same. (see http://stackoverflow.com/questions/
        29596797/can-the-return-value-of-float-as-int-be-used-to-
        compare-float-in-cuda)

    */

    // I am referencing M. Harris' slides
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    // If I have time, I will try to write a better optimized version of this.  

    // Set up shared data
    extern __shared__ float sdata[];

    // Load one element from global to shared memory with each thread
    unsigned int thread_index = threadIdx.x; // thread index in current block
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
    float max_abs = 0.0;    // max absolute value can't be smaller than 0

    // Allow for records of varying length
    // This works by taking the largest value from each and loading it into 
    // shared data.
    while (i < padded_length) {
        // update the max absolute value
        if (fabsf(out_data[i].x) > max_abs) {
            max_abs = fabsf(out_data[i].x);
        }

        // update with grid stride
        i += blockDim.x * gridDim.x;
    }
    sdata[thread_index] = max_abs;  // Store to shared memory

    // Sync threads before doing the reduction max
    __syncthreads(); 

    // Do the reduction in shared memory per block
    // This follows reduction 4 from M. Harris' slide show.
    // It iterates down from 1/2 the block dimension in powers of 2 and stores
    // the result to indices in the first half of the next smallest power of 2.
    // The max values are always organized in the front of the array, not alternating
    // like in the naive approach.  This avoids bank conflicts and increases speed.  
    for(uint s = blockDim.x/2; s > 0; s >>= 1) {
        if (thread_index < s) {
            sdata[thread_index] = max(sdata[thread_index],sdata[thread_index + s]);
        }
        __syncthreads();
    }

    // write the result for this to the global memory
    // Need to use atomic max to check across all blocks being processed on the GPU.
    if (thread_index == 0) {
        atomicMax(max_abs_val, sdata[0]);
    }

}

__global__
void
cudaDivideKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the division kernel. Divide all
    data by the value pointed to by max_abs_val. 

    This kernel should be quite short.
    */

    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    float scaling = 1.0 / *max_abs_val; 

    while (thread_index < padded_length) { 
        // scale each value.  Don't need y.
        out_data[thread_index].x *= scaling;

        // Update the thread index with a grid stride
        thread_index += gridDim.x * blockDim.x;
    }
}


void cudaCallProdScaleKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const hipfftComplex *raw_data,
        const hipfftComplex *impulse_v,
        hipfftComplex *out_data,
        const unsigned int padded_length) {
        
    /* TODO: Call the element-wise product and scaling kernel. */

    cudaProdScaleKernel<<<blocks, threadsPerBlock>>>(raw_data, impulse_v, out_data, padded_length);
}

void cudaCallMaximumKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        

    /* TODO 2: Call the max-finding kernel. */

    cudaMaximumKernel<<<blocks, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(out_data, max_abs_val, padded_length);
}


void cudaCallDivideKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        
    /* TODO 2: Call the division kernel. */

    cudaDivideKernel<<<blocks, threadsPerBlock>>>(out_data, max_abs_val, padded_length);
}
